#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <cuda_device_runtime_api.h>
__global__ void childKernel(){
    if(threadIdx.x == 0 && blockIdx.x == 0){
        printf("hello from childKernel\n");
    }
}


__global__ void parentKernel() { 
    childKernel<<<600000, 64>>>();

    auto ret = hipDeviceSynchronize();
    if(ret != hipSuccess){
        printf("CudaStreamSynchronize failed with %i",ret);
    }
    printf("done\n");
}


int main() {
  hipGraph_t graph;
  hipGraphCreate(&graph,0);
  hipGraphNode_t node;
  hipKernelNodeParams params;
  params.func = (void*) parentKernel;
  params.extra = nullptr;
  params.gridDim = dim3(1);
  params.blockDim = dim3(1);
  params.sharedMemBytes = 0;
  params.kernelParams = nullptr;
  hipGraphAddKernelNode(&node,graph,nullptr,0,&params);

  hipGraphExec_t instance;
  hipGraphInstantiate(&instance,graph,nullptr,nullptr,0);
  
  hipStream_t myStream;
  hipStreamCreate(&myStream);

  for(int i = 0; i < 100000; ++i){
    hipGraphLaunch(instance,myStream);
    auto err = hipStreamSynchronize(myStream);
    if (err != hipSuccess) {
      printf("CUDA Error %d occured\n", err);
      break;
    }

  }
  hipGraphExecDestroy(instance);

  hipGraphDestroy(graph);
  hipStreamDestroy(myStream);

  return 0;
}
